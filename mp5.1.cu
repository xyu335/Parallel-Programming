#include "hip/hip_runtime.h"
// MP Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void total(float *input, float *output, int len) {
  //@@ Load a segment of the input vector into shared memory
  //@@ Traverse the reduction tree
  //@@ Write the computed sum of the block to the output vector at the
  //@@ correct index
  
  __shared__ float Ms[BLOCK_SIZE * 2];
  int tx = threadIdx.x;
  int bx = blockDim.x * blockIdx.x;
  // original sequence
  // len = 256
  int start = bx * 2 + tx * 2; 
  Ms[2 * tx] = start < len ? input[start] : 0;
  Ms[2 * tx + 1] = (start + 1) < len ? input[start + 1] : 0;
  
  // iterate
  unsigned int stride = BLOCK_SIZE;
  for (; stride >= 1; stride /= 2)
  {
    __syncthreads();
    /*if (stride == 512 && tx == 0) 
    {
      printf("len %d \n", len);
      for (int i = 0; i < 512; ++i)
        printf("%dth: %f\n", Ms[i]);
    }*/
    if (tx < stride)
    {
      Ms[tx] = Ms[tx] + Ms[tx + stride];
    }
  }
  
  if (tx == 0) 
  {
    printf("the value: %f", Ms[0]);
    output[blockIdx.x] = Ms[0];
  }
  return;
}

int main(int argc, char **argv) {
  int ii;
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numInputElements;  // number of elements in the input list
  int numOutputElements; // number of elements in the output list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

  // BLOCK_SIZE * 2
  numOutputElements = numInputElements / (BLOCK_SIZE << 1);
  if (numInputElements % (BLOCK_SIZE << 1)) {
    numOutputElements++;
  }
  hostOutput = (float *)malloc(numOutputElements * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numInputElements);
  wbLog(TRACE, "The number of output elements in the input is ",
        numOutputElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceInput, sizeof(float) * numInputElements);
  hipMalloc((void **)&deviceOutput, sizeof(float) * numOutputElements);

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, sizeof(float) * numInputElements, hipMemcpyHostToDevice);
  
  wbTime_stop(GPU, "Copying input memory to the GPU.");
  //@@ Initialize the grid and block dimensions here
  dim3 blockDim(BLOCK_SIZE, 1, 1);
  dim3 gridDim(numOutputElements, 1, 1);
  
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  total<<<gridDim, blockDim>>>(deviceInput, deviceOutput, numInputElements);
  
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);
  
  wbTime_stop(Copy, "Copying output memory to the CPU");

  /********************************************************************
   * Reduce output vector on the host
   * NOTE: One could also perform the reduction of the output vector
   * recursively and support any size input. For simplicity, we do not
   * require that for this lab.
   ********************************************************************/
  printf("value: %f for main \n", hostOutput[0]);
  for (ii = 1; ii < numOutputElements; ii++) {
    printf("result: %dth %f\n", ii, hostOutput[ii]);
    hostOutput[0] += hostOutput[ii];
  }

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceOutput);
  hipFree(deviceInput);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}

